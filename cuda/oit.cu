#include "hip/hip_runtime.h"

#include <map>
#include <string>
#include <set>
#include <vector>
#include <list>
#include <pyarlib/includegl.h>
#include <pyarlib/util.h>
#include <inttypes.h>
#include <assert.h>

#include "oit.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <cudaGL.h>
//#include <cudpp.h>
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check
#include <helper_cuda_gl.h>      // helper functions for CUDA/GL interop
#include <hip/hip_vector_types.h>


inline __host__ __device__ float4 operator+(float4 a, float4 b)
{
    return make_float4(a.x + b.x, a.y + b.y, a.z + b.z,  a.w + b.w);
}
inline __host__ __device__ float4 operator*(float4 a, float s)
{
    return make_float4(a.x * s, a.y * s, a.z * s, a.w * s);
}
inline __device__ float fract(float x)
{
	return x - floor(x);
}
__device__ float4 floatToRGBA8(float x)
{
	union { float f; unsigned int i; } tmp;
	tmp.f = x;
	unsigned int i = tmp.i;
	return make_float4(
		((float)(i>>24))/255.0f,
		((float)((i>>16)&0xFF))/255.0f,
		((float)((i>>8)&0xFF))/255.0f,
		((float)(i & 0xFF))/255.0f
		);
}


template<int A> struct Log2 {
	enum
	{
		value = Log2<(A >> 1)>::value + 1
	};
};
template<> struct Log2<1> {enum{value = 0};};
//Usage:
//uint n=Log2<16u>::value;

template <int A, int B>
struct Ceil
{
	enum
	{
		value = (A / B) + (A % B == 0 ? 0 : 1)
	};
};

template<size_t MAX_FRAGS>
__global__ void kernelLinkedLists(unsigned int* headPtrs, unsigned int* nextPtrs, float* data, uchar4* framebuffer, int stride, int n)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= n)
		return;
	
	float2 frags[MAX_FRAGS];
	
	int count = 0;
	
	int node = headPtrs[index];
	while (node != 0 && count < MAX_FRAGS)
	{
		frags[count].x = data[node * stride];
		frags[count].y = data[node * stride + 1];
		++count;
		node = nextPtrs[node];
	}
	
	for (int j = 1; j < count; ++j)
	{
		float2 key = frags[j];
		int i = j - 1;
		while (i >= 0 && frags[i].y > key.y)
		{
			frags[i+1] = frags[i];
			--i;
		}
		frags[i+1] = key;
	}
	
	float4 fragColour = {1.0f, 1.0f, 1.0f, 1.0f};
	for (int i = 0; i < count; ++i)
	{
		float4 col = floatToRGBA8(frags[count-i-1].x);
		fragColour = fragColour * (1.0 - col.w) + col * col.w;
	}
	
	fragColour = fragColour * 255;
	
	framebuffer[index] = make_uchar4(fragColour.x, fragColour.y, fragColour.z, 255);
	
	//unsigned char complexity = count;
	//framebuffer[index] = make_uchar4(complexity, complexity, complexity, 255);
	//framebuffer[index] = debug;
	//framebuffer[index] = make_uchar4(threadIndex%800, threadIndex/800, 0, 255);
}

#define FRAGS(x) frags[x]
#define LFB_FRAG_DEPTH(x) (x).y

template<size_t MAX_FRAGS>
__device__ void merge(float2 frags[], int step, int a, int b, int c)
{
	float2 leftArray[MAX_FRAGS/2];

	int i;
	for (i = 0; i < step; ++i)
		leftArray[i] = FRAGS(a+i);

	i = 0;
	int j = 0;
	for (int k = a; k < c; ++k)
	{
		if (b+j >= c || (i < step && LFB_FRAG_DEPTH(leftArray[i]) < LFB_FRAG_DEPTH(FRAGS(b+j))))
			FRAGS(k) = leftArray[i++];
		else
			FRAGS(k) = FRAGS(b+j++);
	}
}

template<size_t MAX_FRAGS>
__device__ void sort_merge(float2 frags[], int fragCount)
{
	int n = fragCount;
	int step = 1;
	while (step <= n)
	{
		int i = 0;
		while (i < n - step)
		{
			merge<MAX_FRAGS>(frags, step, i, i + step, min(i + step + step, n));
			i += 2 * step;
		}
		step *= 2;
	}
}

template<size_t MAX_FRAGS>
__global__ void kernelLinkedListsMerge(unsigned int* headPtrs, unsigned int* nextPtrs, float* data, uchar4* framebuffer, int stride, int n)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= n)
		return;
	
	float2 frags[MAX_FRAGS];
	
	int count = 0;
	
	int node = headPtrs[index];
	while (node != 0 && count < MAX_FRAGS)
	{
		frags[count].x = data[node * stride];
		frags[count].y = data[node * stride + 1];
		++count;
		node = nextPtrs[node];
	}
	
	if (count >= 32)
		sort_merge<MAX_FRAGS>(frags, count);
	else
	{
		for (int j = 1; j < count; ++j)
		{
			float2 key = frags[j];
			int i = j - 1;
			while (i >= 0 && frags[i].y > key.y)
			{
				frags[i+1] = frags[i];
				--i;
			}
			frags[i+1] = key;
		}
	}
	
	float4 fragColour = {1.0f, 1.0f, 1.0f, 1.0f};
	for (int i = 0; i < count; ++i)
	{
		float4 col = floatToRGBA8(frags[count-i-1].x);
		fragColour = fragColour * (1.0 - col.w) + col * col.w;
	}
	
	fragColour = fragColour * 255;
	
	framebuffer[index] = make_uchar4(fragColour.x, fragColour.y, fragColour.z, 255);
	
	//unsigned char complexity = count;
	//framebuffer[index] = make_uchar4(complexity, complexity, complexity, 255);
	//framebuffer[index] = debug;
	//framebuffer[index] = make_uchar4(threadIndex%800, threadIndex/800, 0, 255);
}

#define NUM_REGISTERS 32

__device__ void sortInRegisters(float2* frags, int count)
{
	/*
	//insertion sort into registers. OH WAIT NO. CUDA IS RETARDED AND WON'T UNROLL THE LOOPS
	#pragma unroll
	for (int i = 0; i < N; ++i)
	{
		if (i < count)
		{
			int j;
			float2 next = frags[i];
			#pragma unroll
			for (j = i; j > 0; --j)
				if (next.y < registers[j-1].y)
					registers[j] = registers[j-1];
			registers[j] = next;
		}
	}
	*/
	
	float2 tmp;
	#define SWAP(a, b) \
		if (frag##a.y > frag##b.y) {tmp = frag##a; frag##a = frag##b; frag##b = tmp;}
	
	#define N NUM_REGISTERS
	
	//BEGIN GENERATED
#if N > 0
float2 frag0,frag1,frag2,frag3,frag4,frag5,frag6,frag7;
#endif
#if N > 8
float2 frag8,frag9,frag10,frag11,frag12,frag13,frag14,frag15;
#endif
#if N > 16
float2 frag16,frag17,frag18,frag19,frag20,frag21,frag22,frag23,frag24,frag25,frag26,frag27,frag28,frag29,frag30,frag31;
#endif
#if N > 0
if (count > 0) frag0 = frags[0];
if (count > 1) frag1 = frags[1];
if (count > 2) frag2 = frags[2];
if (count > 3) frag3 = frags[3];
if (count > 4) frag4 = frags[4];
if (count > 5) frag5 = frags[5];
if (count > 6) frag6 = frags[6];
if (count > 7) frag7 = frags[7];
#endif
#if N > 8
if (count > 8) frag8 = frags[8];
if (count > 9) frag9 = frags[9];
if (count > 10) frag10 = frags[10];
if (count > 11) frag11 = frags[11];
if (count > 12) frag12 = frags[12];
if (count > 13) frag13 = frags[13];
if (count > 14) frag14 = frags[14];
if (count > 15) frag15 = frags[15];
#endif
#if N > 16
if (count > 16) frag16 = frags[16];
if (count > 17) frag17 = frags[17];
if (count > 18) frag18 = frags[18];
if (count > 19) frag19 = frags[19];
if (count > 20) frag20 = frags[20];
if (count > 21) frag21 = frags[21];
if (count > 22) frag22 = frags[22];
if (count > 23) frag23 = frags[23];
if (count > 24) frag24 = frags[24];
if (count > 25) frag25 = frags[25];
if (count > 26) frag26 = frags[26];
if (count > 27) frag27 = frags[27];
if (count > 28) frag28 = frags[28];
if (count > 29) frag29 = frags[29];
if (count > 30) frag30 = frags[30];
if (count > 31) frag31 = frags[31];
#endif

#if N > 0
if (count > 0) {
if (count > 1) {SWAP(0, 1);
if (count > 2) {SWAP(1, 2);SWAP(0, 1);
if (count > 3) {SWAP(2, 3);SWAP(1, 2);SWAP(0, 1);
if (count > 4) {SWAP(3, 4);SWAP(2, 3);SWAP(1, 2);SWAP(0, 1);
if (count > 5) {SWAP(4, 5);SWAP(3, 4);SWAP(2, 3);SWAP(1, 2);SWAP(0, 1);
if (count > 6) {SWAP(5, 6);SWAP(4, 5);SWAP(3, 4);SWAP(2, 3);SWAP(1, 2);SWAP(0, 1);
if (count > 7) {SWAP(6, 7);SWAP(5, 6);SWAP(4, 5);SWAP(3, 4);SWAP(2, 3);SWAP(1, 2);SWAP(0, 1);
#if N > 8
if (count > 8) {SWAP(7, 8);SWAP(6, 7);SWAP(5, 6);SWAP(4, 5);SWAP(3, 4);SWAP(2, 3);SWAP(1, 2);SWAP(0, 1);
if (count > 9) {SWAP(8, 9);SWAP(7, 8);SWAP(6, 7);SWAP(5, 6);SWAP(4, 5);SWAP(3, 4);SWAP(2, 3);SWAP(1, 2);SWAP(0, 1);
if (count > 10) {SWAP(9, 10);SWAP(8, 9);SWAP(7, 8);SWAP(6, 7);SWAP(5, 6);SWAP(4, 5);SWAP(3, 4);SWAP(2, 3);SWAP(1, 2);SWAP(0, 1);
if (count > 11) {SWAP(10, 11);SWAP(9, 10);SWAP(8, 9);SWAP(7, 8);SWAP(6, 7);SWAP(5, 6);SWAP(4, 5);SWAP(3, 4);SWAP(2, 3);SWAP(1, 2);SWAP(0, 1);
if (count > 12) {SWAP(11, 12);SWAP(10, 11);SWAP(9, 10);SWAP(8, 9);SWAP(7, 8);SWAP(6, 7);SWAP(5, 6);SWAP(4, 5);SWAP(3, 4);SWAP(2, 3);SWAP(1, 2);SWAP(0, 1);
if (count > 13) {SWAP(12, 13);SWAP(11, 12);SWAP(10, 11);SWAP(9, 10);SWAP(8, 9);SWAP(7, 8);SWAP(6, 7);SWAP(5, 6);SWAP(4, 5);SWAP(3, 4);SWAP(2, 3);SWAP(1, 2);SWAP(0, 1);
if (count > 14) {SWAP(13, 14);SWAP(12, 13);SWAP(11, 12);SWAP(10, 11);SWAP(9, 10);SWAP(8, 9);SWAP(7, 8);SWAP(6, 7);SWAP(5, 6);SWAP(4, 5);SWAP(3, 4);SWAP(2, 3);SWAP(1, 2);SWAP(0, 1);
if (count > 15) {SWAP(14, 15);SWAP(13, 14);SWAP(12, 13);SWAP(11, 12);SWAP(10, 11);SWAP(9, 10);SWAP(8, 9);SWAP(7, 8);SWAP(6, 7);SWAP(5, 6);SWAP(4, 5);SWAP(3, 4);SWAP(2, 3);SWAP(1, 2);SWAP(0, 1);
#if N > 16
if (count > 16) {SWAP(15, 16);SWAP(14, 15);SWAP(13, 14);SWAP(12, 13);SWAP(11, 12);SWAP(10, 11);SWAP(9, 10);SWAP(8, 9);SWAP(7, 8);SWAP(6, 7);SWAP(5, 6);SWAP(4, 5);SWAP(3, 4);SWAP(2, 3);SWAP(1, 2);SWAP(0, 1);
if (count > 17) {SWAP(16, 17);SWAP(15, 16);SWAP(14, 15);SWAP(13, 14);SWAP(12, 13);SWAP(11, 12);SWAP(10, 11);SWAP(9, 10);SWAP(8, 9);SWAP(7, 8);SWAP(6, 7);SWAP(5, 6);SWAP(4, 5);SWAP(3, 4);SWAP(2, 3);SWAP(1, 2);SWAP(0, 1);
if (count > 18) {SWAP(17, 18);SWAP(16, 17);SWAP(15, 16);SWAP(14, 15);SWAP(13, 14);SWAP(12, 13);SWAP(11, 12);SWAP(10, 11);SWAP(9, 10);SWAP(8, 9);SWAP(7, 8);SWAP(6, 7);SWAP(5, 6);SWAP(4, 5);SWAP(3, 4);SWAP(2, 3);SWAP(1, 2);SWAP(0, 1);
if (count > 19) {SWAP(18, 19);SWAP(17, 18);SWAP(16, 17);SWAP(15, 16);SWAP(14, 15);SWAP(13, 14);SWAP(12, 13);SWAP(11, 12);SWAP(10, 11);SWAP(9, 10);SWAP(8, 9);SWAP(7, 8);SWAP(6, 7);SWAP(5, 6);SWAP(4, 5);SWAP(3, 4);SWAP(2, 3);SWAP(1, 2);SWAP(0, 1);
if (count > 20) {SWAP(19, 20);SWAP(18, 19);SWAP(17, 18);SWAP(16, 17);SWAP(15, 16);SWAP(14, 15);SWAP(13, 14);SWAP(12, 13);SWAP(11, 12);SWAP(10, 11);SWAP(9, 10);SWAP(8, 9);SWAP(7, 8);SWAP(6, 7);SWAP(5, 6);SWAP(4, 5);SWAP(3, 4);SWAP(2, 3);SWAP(1, 2);SWAP(0, 1);
if (count > 21) {SWAP(20, 21);SWAP(19, 20);SWAP(18, 19);SWAP(17, 18);SWAP(16, 17);SWAP(15, 16);SWAP(14, 15);SWAP(13, 14);SWAP(12, 13);SWAP(11, 12);SWAP(10, 11);SWAP(9, 10);SWAP(8, 9);SWAP(7, 8);SWAP(6, 7);SWAP(5, 6);SWAP(4, 5);SWAP(3, 4);SWAP(2, 3);SWAP(1, 2);SWAP(0, 1);
if (count > 22) {SWAP(21, 22);SWAP(20, 21);SWAP(19, 20);SWAP(18, 19);SWAP(17, 18);SWAP(16, 17);SWAP(15, 16);SWAP(14, 15);SWAP(13, 14);SWAP(12, 13);SWAP(11, 12);SWAP(10, 11);SWAP(9, 10);SWAP(8, 9);SWAP(7, 8);SWAP(6, 7);SWAP(5, 6);SWAP(4, 5);SWAP(3, 4);SWAP(2, 3);SWAP(1, 2);SWAP(0, 1);
if (count > 23) {SWAP(22, 23);SWAP(21, 22);SWAP(20, 21);SWAP(19, 20);SWAP(18, 19);SWAP(17, 18);SWAP(16, 17);SWAP(15, 16);SWAP(14, 15);SWAP(13, 14);SWAP(12, 13);SWAP(11, 12);SWAP(10, 11);SWAP(9, 10);SWAP(8, 9);SWAP(7, 8);SWAP(6, 7);SWAP(5, 6);SWAP(4, 5);SWAP(3, 4);SWAP(2, 3);SWAP(1, 2);SWAP(0, 1);
if (count > 24) {SWAP(23, 24);SWAP(22, 23);SWAP(21, 22);SWAP(20, 21);SWAP(19, 20);SWAP(18, 19);SWAP(17, 18);SWAP(16, 17);SWAP(15, 16);SWAP(14, 15);SWAP(13, 14);SWAP(12, 13);SWAP(11, 12);SWAP(10, 11);SWAP(9, 10);SWAP(8, 9);SWAP(7, 8);SWAP(6, 7);SWAP(5, 6);SWAP(4, 5);SWAP(3, 4);SWAP(2, 3);SWAP(1, 2);SWAP(0, 1);
if (count > 25) {SWAP(24, 25);SWAP(23, 24);SWAP(22, 23);SWAP(21, 22);SWAP(20, 21);SWAP(19, 20);SWAP(18, 19);SWAP(17, 18);SWAP(16, 17);SWAP(15, 16);SWAP(14, 15);SWAP(13, 14);SWAP(12, 13);SWAP(11, 12);SWAP(10, 11);SWAP(9, 10);SWAP(8, 9);SWAP(7, 8);SWAP(6, 7);SWAP(5, 6);SWAP(4, 5);SWAP(3, 4);SWAP(2, 3);SWAP(1, 2);SWAP(0, 1);
if (count > 26) {SWAP(25, 26);SWAP(24, 25);SWAP(23, 24);SWAP(22, 23);SWAP(21, 22);SWAP(20, 21);SWAP(19, 20);SWAP(18, 19);SWAP(17, 18);SWAP(16, 17);SWAP(15, 16);SWAP(14, 15);SWAP(13, 14);SWAP(12, 13);SWAP(11, 12);SWAP(10, 11);SWAP(9, 10);SWAP(8, 9);SWAP(7, 8);SWAP(6, 7);SWAP(5, 6);SWAP(4, 5);SWAP(3, 4);SWAP(2, 3);SWAP(1, 2);SWAP(0, 1);
if (count > 27) {SWAP(26, 27);SWAP(25, 26);SWAP(24, 25);SWAP(23, 24);SWAP(22, 23);SWAP(21, 22);SWAP(20, 21);SWAP(19, 20);SWAP(18, 19);SWAP(17, 18);SWAP(16, 17);SWAP(15, 16);SWAP(14, 15);SWAP(13, 14);SWAP(12, 13);SWAP(11, 12);SWAP(10, 11);SWAP(9, 10);SWAP(8, 9);SWAP(7, 8);SWAP(6, 7);SWAP(5, 6);SWAP(4, 5);SWAP(3, 4);SWAP(2, 3);SWAP(1, 2);SWAP(0, 1);
if (count > 28) {SWAP(27, 28);SWAP(26, 27);SWAP(25, 26);SWAP(24, 25);SWAP(23, 24);SWAP(22, 23);SWAP(21, 22);SWAP(20, 21);SWAP(19, 20);SWAP(18, 19);SWAP(17, 18);SWAP(16, 17);SWAP(15, 16);SWAP(14, 15);SWAP(13, 14);SWAP(12, 13);SWAP(11, 12);SWAP(10, 11);SWAP(9, 10);SWAP(8, 9);SWAP(7, 8);SWAP(6, 7);SWAP(5, 6);SWAP(4, 5);SWAP(3, 4);SWAP(2, 3);SWAP(1, 2);SWAP(0, 1);
if (count > 29) {SWAP(28, 29);SWAP(27, 28);SWAP(26, 27);SWAP(25, 26);SWAP(24, 25);SWAP(23, 24);SWAP(22, 23);SWAP(21, 22);SWAP(20, 21);SWAP(19, 20);SWAP(18, 19);SWAP(17, 18);SWAP(16, 17);SWAP(15, 16);SWAP(14, 15);SWAP(13, 14);SWAP(12, 13);SWAP(11, 12);SWAP(10, 11);SWAP(9, 10);SWAP(8, 9);SWAP(7, 8);SWAP(6, 7);SWAP(5, 6);SWAP(4, 5);SWAP(3, 4);SWAP(2, 3);SWAP(1, 2);SWAP(0, 1);
if (count > 30) {SWAP(29, 30);SWAP(28, 29);SWAP(27, 28);SWAP(26, 27);SWAP(25, 26);SWAP(24, 25);SWAP(23, 24);SWAP(22, 23);SWAP(21, 22);SWAP(20, 21);SWAP(19, 20);SWAP(18, 19);SWAP(17, 18);SWAP(16, 17);SWAP(15, 16);SWAP(14, 15);SWAP(13, 14);SWAP(12, 13);SWAP(11, 12);SWAP(10, 11);SWAP(9, 10);SWAP(8, 9);SWAP(7, 8);SWAP(6, 7);SWAP(5, 6);SWAP(4, 5);SWAP(3, 4);SWAP(2, 3);SWAP(1, 2);SWAP(0, 1);
if (count > 31) {SWAP(30, 31);SWAP(29, 30);SWAP(28, 29);SWAP(27, 28);SWAP(26, 27);SWAP(25, 26);SWAP(24, 25);SWAP(23, 24);SWAP(22, 23);SWAP(21, 22);SWAP(20, 21);SWAP(19, 20);SWAP(18, 19);SWAP(17, 18);SWAP(16, 17);SWAP(15, 16);SWAP(14, 15);SWAP(13, 14);SWAP(12, 13);SWAP(11, 12);SWAP(10, 11);SWAP(9, 10);SWAP(8, 9);SWAP(7, 8);SWAP(6, 7);SWAP(5, 6);SWAP(4, 5);SWAP(3, 4);SWAP(2, 3);SWAP(1, 2);SWAP(0, 1);
}}}}}}}}}}}}}}}}
#endif
}}}}}}}}
#endif
}}}}}}}}
#endif

#if N > 0
if (count > 0) frags[0] = frag0;
if (count > 1) frags[1] = frag1;
if (count > 2) frags[2] = frag2;
if (count > 3) frags[3] = frag3;
if (count > 4) frags[4] = frag4;
if (count > 5) frags[5] = frag5;
if (count > 6) frags[6] = frag6;
if (count > 7) frags[7] = frag7;
#endif
#if N > 8
if (count > 8) frags[8] = frag8;
if (count > 9) frags[9] = frag9;
if (count > 10) frags[10] = frag10;
if (count > 11) frags[11] = frag11;
if (count > 12) frags[12] = frag12;
if (count > 13) frags[13] = frag13;
if (count > 14) frags[14] = frag14;
if (count > 15) frags[15] = frag15;
#endif
#if N > 16
if (count > 16) frags[16] = frag16;
if (count > 17) frags[17] = frag17;
if (count > 18) frags[18] = frag18;
if (count > 19) frags[19] = frag19;
if (count > 20) frags[20] = frag20;
if (count > 21) frags[21] = frag21;
if (count > 22) frags[22] = frag22;
if (count > 23) frags[23] = frag23;
if (count > 24) frags[24] = frag24;
if (count > 25) frags[25] = frag25;
if (count > 26) frags[26] = frag26;
if (count > 27) frags[27] = frag27;
if (count > 28) frags[28] = frag28;
if (count > 29) frags[29] = frag29;
if (count > 30) frags[30] = frag30;
if (count > 31) frags[31] = frag31;
#endif

	//END GENERATED
}


template<size_t MAX_FRAGS>
__global__ void kernelLinkedListsRegisters(unsigned int* headPtrs, unsigned int* nextPtrs, float* data, uchar4* framebuffer, int stride, int n)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= n)
		return;
	
	float2 frags[MAX_FRAGS];
	
	int count = 0;
	
	int node = headPtrs[index];
	while (node != 0 && count < MAX_FRAGS)
	{
		frags[count].x = data[node * stride];
		frags[count].y = data[node * stride + 1];
		++count;
		node = nextPtrs[node];
	}
	
	//sort blocks in registers
	for (int i = 0; i < count; i += NUM_REGISTERS)
		sortInRegisters(frags + i, min(count - i, NUM_REGISTERS));
	//sortInRegisters<NUM_REGISTERS>(frags, NUM_REGISTERS);
	
	const int MERGE_SIZE = Ceil<MAX_FRAGS, NUM_REGISTERS>::value;
	
	//begin min-finding (actually since we want reverse order it's max-finding)
	int next[MERGE_SIZE];
	#pragma unroll
	for (int i = 0; i < MERGE_SIZE; ++i)
		next[i] = min(count, (i + 1) * NUM_REGISTERS) - 1;
	
	float4 fragColour = {1.0f, 1.0f, 1.0f, 1.0f};
	for (int i = 0; i < count; ++i)
	{
		#if 1
		
		int n; //I'll assume n *will* be set by the end of the loop
		float2 f;
		f.y = 0.0;
		#pragma unroll
		for (int j = 0; j < MERGE_SIZE; ++j)
		{
			if (next[j] >= j * NUM_REGISTERS)
			{
				if (frags[next[j]].y > f.y)
				{
					f = frags[next[j]];
					n = j;
				}
			}
		}
		
		#pragma unroll
		for (int j = 0; j < MERGE_SIZE; ++j)
			if (n == j)
				--next[j];
		
		#else
		
		float2 f = frags[i];
		
		#endif
		
		float4 col = floatToRGBA8(f.x);
		fragColour = fragColour * (1.0 - col.w) + col * col.w;
	}
	
	fragColour = fragColour * 255;
	
	framebuffer[index] = make_uchar4(fragColour.x, fragColour.y, fragColour.z, 255);
	
	//unsigned char complexity = count;
	//framebuffer[index] = make_uchar4(complexity, complexity, complexity, 255);
	//framebuffer[index] = debug;
	//framebuffer[index] = make_uchar4(threadIndex%800, threadIndex/800, 0, 255);
}

template<size_t NFRAGS>
__global__ void kernelLinearizedParallel(float* data, uchar4* out, int n)
{
	//*out = make_uchar4(n, n, 255, 255);
	//return;
	
	__shared__ float2 frags[NFRAGS];
	int i = threadIdx.x;
	
	//load the data
	if (i < n)
	{
		frags[i].x = data[i*2+0];
		frags[i].y = data[i*2+1];
	}
	else
	{
		frags[i].x = 0.0f;
		frags[i].y = 999.0f;
	}
	__syncthreads();

	int logn = Log2<NFRAGS>::value;
	float2 tmp;
	#define SWAPB(a, b) \
		if (a.y > b.y) {tmp = a; a = b; b = tmp;}

	//sort with bitonic sorting network
	for (int lk = 1; lk <= logn; ++lk)
	{
		int k = 1<<lk;
		for (int lj = lk-1; lj >= 0; --lj)
		{
			int j = 1<<lj;
			int ixj=i^j;
			if (ixj > i)
			{
				if ((i&k)==0) SWAPB(frags[i],frags[ixj])
				if ((i&k)!=0) SWAPB(frags[ixj],frags[i])
			}
			__syncthreads();
		}
	}
	
	//blend in pairs
	for (int lk = 0; lk < logn; ++lk)
	{
		int k = 1<<lk;
		if (i % k == 0)
		{
			uchar4& col = *reinterpret_cast<uchar4*>(&frags[i].x);
			uchar4 col2 = *reinterpret_cast<uchar4*>(&frags[i+k].x);
			float a = col.w / 255.0f;
			col2.x = col.x * a + col2.x * (1.0f - a);
			col2.y = col.y * a + col2.y * (1.0f - a);
			col2.z = col.z * a + col2.z * (1.0f - a);
			col2.w = 1.0f - (1.0f - a) * (1.0f - col2.w / 255.0f);
			frags[i].x = *reinterpret_cast<float*>(&col2);
		}
		__syncthreads();
	}
	
	//write the result
	if (i == 0)
	{
		*out = *reinterpret_cast<uchar4*>(&frags[0].x);
	}
}

template<size_t NFRAGS, size_t TPB>
__global__ void kernelLinearizedParallelSpawner(unsigned int* offsets, float* data, uchar4* framebuffer, int stride, int n)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= n)
		return;

	int offset = 0;
	if (index > 0)
		offset = offsets[index-1];
	int count = min((int)NFRAGS, (int)(offsets[index] - offset));
	
	kernelLinearizedParallel<NFRAGS><<<1, NFRAGS, NFRAGS*sizeof(float2)>>>(data + offset, framebuffer + index, count);
}

template<size_t MAX_FRAGS, size_t TPB>
__global__ void kernelLinearized(unsigned int* offsets, float* data, uchar4* framebuffer, int stride, int n)
{
	int index = blockIdx.x * TPB + threadIdx.x;

	__shared__ float2 temp[TPB];
	float2 frags[MAX_FRAGS];
	int fo = threadIdx.x * MAX_FRAGS;
	
	int count = 0;
	
	float4 fragColour = {1.0f, 1.0f, 1.0f, 1.0f};
	
	//read this: http://cuda-programming.blogspot.com.au/2013/02/bank-conflicts-in-shared-memory-in-cuda.html
	
	for (int i = 0; i < TPB; ++i) //i is the thread in this warp we're reading stuff for
	{
		int p = blockIdx.x * TPB + i; //p is the pixel index
		if (p < n)
		{
			for (int j = 0; j < MAX_FRAGS/TPB; ++j) //j is the element in i's array we're reading
			{
				//read the pixel's offset and count
				int o = 0;
				if (p > 0)
					o = offsets[p-1];
				int c = min((int)MAX_FRAGS, (int)(offsets[p] - o));
				
				//if there's data for us to load, load it
				int f = (j*TPB+threadIdx.x);
				if (f < c)
				{
					temp[threadIdx.x].x = data[(o + f) * stride + 0];
					temp[threadIdx.x].y = data[(o + f) * stride + 1];
				}
				
				__syncthreads();
				
				if (i == threadIdx.x)
				{
					count = c;
					for (int k = 0; k < TPB && j*TPB+k < c; ++k)
						frags[j*TPB+k] = temp[k];
				}
				
				__syncthreads();
			}
		}
	}
	
	if (index >= n)
		return;

	for (int j = 1; j < count; ++j)
	{
		float2 key = frags[fo+j];
		int i = j - 1;
		while (i >= 0 && frags[fo+i].y > key.y)
		{
			frags[fo+i+1] = frags[fo+i];
			--i;
		}
		frags[fo+i+1] = key;
	}
	
	for (int i = 0; i < count; ++i)
	{
		float4 col = floatToRGBA8(frags[fo+count-i-1].x);
		fragColour = fragColour * (1.0 - col.w) + col * col.w;
	}
	
	fragColour = fragColour * 255;
	
	framebuffer[index] = make_uchar4(fragColour.x, fragColour.y, fragColour.z, 255);
	
	//unsigned char complexity = count;
	//framebuffer[index] = make_uchar4(complexity, complexity, complexity, 255);
	//framebuffer[index] = debug;
	//framebuffer[index] = make_uchar4(threadIndex%800, threadIndex/800, 0, 255);
}

#define CHECK_CUDA_ERROR _checkCudaError(__FILE__, __LINE__)

bool _checkCudaError(const char* file, int line)
{
	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
	{
		printf("CUDA Error %s:%i: %s\n", file, line, hipGetErrorString(err));
		return true;
	}
	return false;
}

class CUDAGLBuffer {
private:
	bool mapped;
	GLuint buffer;
	hipGraphicsResource_t resource;
	static std::map<GLuint, CUDAGLBuffer*> cache;
public:
	CUDAGLBuffer(GLuint buffer) : buffer(buffer), resource(NULL)
	{
		printf("Created CUDA mapping for buffer %i\n", buffer);
		mapped = false;
		registerBuffer();
	}
	virtual ~CUDAGLBuffer()
	{
		unregisterBuffer();
	}
	void registerBuffer()
	{
		resource = NULL;
		checkCudaErrors(hipGraphicsGLRegisterBuffer(&resource, buffer, hipGraphicsRegisterFlagsNone));
		assert(resource != NULL);
	}
	void unregisterBuffer()
	{
		checkCudaErrors(hipGraphicsUnregisterResource(resource));
		resource = NULL;
	}
	void* map()
	{
		if (!resource)
			registerBuffer();
	
		size_t size = 0;
		void* ptr = NULL;
		checkCudaErrors(hipGraphicsMapResources(1, &resource, 0));
		checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&ptr, &size, resource));
		mapped = true;
		
		int actualSize;
		glBindBuffer(GL_ARRAY_BUFFER, buffer);
		glGetBufferParameteriv(GL_ARRAY_BUFFER, GL_BUFFER_SIZE, &actualSize);
		glBindBuffer(GL_ARRAY_BUFFER, 0);
		
		if (size != actualSize) //re-register if size has changed... uuuuurgh!
		{
			printf("CUDA RE-REGISTER %i\n", buffer);
			unmap(); unregisterBuffer(); registerBuffer();
			checkCudaErrors(hipGraphicsMapResources(1, &resource, 0));
			checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&ptr, &size, resource));
			mapped = true;
			assert(size == actualSize);
		}
		
		return ptr;
	}
	void unmap()
	{
		if (!mapped)
			return;
		
		checkCudaErrors(hipGraphicsUnmapResources(1, &resource, 0));
		mapped = false;
	}
	static struct Getter {
		void* operator[](GLuint i)
		{
			if (cache.find(i) == cache.end())
				cache[i] = new CUDAGLBuffer(i);
			return cache[i]->map();
		}
	} get;
	static void unmapAll()
	{
		std::map<GLuint, CUDAGLBuffer*>::iterator it;
		for (it = cache.begin(); it != cache.end(); ++it)
			it->second->unmap();
	}
	static void refreshAll()
	{
		std::map<GLuint, CUDAGLBuffer*>::iterator it;
		for (it = cache.begin(); it != cache.end(); ++it)
			it->second->unregisterBuffer();
	}
};
std::map<GLuint, CUDAGLBuffer*> CUDAGLBuffer::cache;
CUDAGLBuffer::Getter CUDAGLBuffer::get;

void refreshCUDABuffers()
{
	CUDAGLBuffer::refreshAll();
	printf("Refreshing buffers\n");
}

bool initCUDA()
{
	static bool hasInit = false;
	if (hasInit)
	{
		printf("Warning: trying to init cuda multiple times\n");
		return true;
	}
	
	int num_devices, device;
	hipGetDeviceCount(&num_devices);
	if (num_devices == 0)
	{
		printf("NO CUDA DEVICES FOUND\n");
		return false;
	}
	int max_multiprocessors = 0, max_device = 0;
	for (device = 0; device < num_devices; device++)
	{
		hipDeviceProp_t properties;
		hipGetDeviceProperties(&properties, device);
		printf("Found CUDA DEVICE %i: %s %i %i\n", device, properties.name, properties.sharedMemPerBlock, properties.multiProcessorCount);
		if (max_multiprocessors < properties.multiProcessorCount) {
			max_multiprocessors = properties.multiProcessorCount;
			max_device = device;
		}
	}

	max_device = 0;

	cudaGLSetGLDevice(0);
	printf("Chose %i\n", max_device);
	hipSetDevice(max_device);
	
	checkCudaErrors(hipDeviceSetCacheConfig(hipFuncCachePreferL1));
	
	hasInit = true;
	return true;
}

#define SWITCH_FRAGS(maxFrags, kernel, ...) \
	switch (maxFrags) \
	{ \
	case 8: kernel<8><<<grid,block>>>(__VA_ARGS__); break; \
	case 16: kernel<16><<<grid,block>>>(__VA_ARGS__); break; \
	case 32: kernel<32><<<grid,block>>>(__VA_ARGS__); break; \
	case 64: kernel<64><<<grid,block>>>(__VA_ARGS__); break; \
	case 128: kernel<128><<<grid,block>>>(__VA_ARGS__); break; \
	case 256: kernel<256><<<grid,block>>>(__VA_ARGS__); break; \
	case 512: kernel<512><<<grid,block>>>(__VA_ARGS__); break; \
	default: success = false; break; \
	} \

bool compositeLinkedLists(int heads, int nexts, int data, int outBufferTexture, int pixels, int maxFrags, bool mergeSort, bool registerSort)
{
	int stride = 2;
	unsigned int* headPtrs = (unsigned int*)CUDAGLBuffer::get[heads];
	unsigned int* nextPtrs = (unsigned int*)CUDAGLBuffer::get[nexts];
	float* dataPtr = (float*)CUDAGLBuffer::get[data];
	uchar4* framebuffer = (uchar4*)CUDAGLBuffer::get[outBufferTexture];
	
	//FIXME: 32 TPB is always faster. doesn't make sense - should then be limited by blocks per SM
	int tpb = 32;
	dim3 grid(ceil(pixels, tpb), 1, 1);
	dim3 block(tpb, 1, 1);
	
	bool success = true;
	
	
	if (registerSort)
	{
		switch (maxFrags)
		{
		case 8: kernelLinkedListsRegisters<8><<<grid,block>>>(headPtrs, nextPtrs, dataPtr, framebuffer, stride, pixels); break;
		case 16: kernelLinkedListsRegisters<16><<<grid,block>>>(headPtrs, nextPtrs, dataPtr, framebuffer, stride, pixels); break;
		case 32: kernelLinkedListsRegisters<32><<<grid,block>>>(headPtrs, nextPtrs, dataPtr, framebuffer, stride, pixels); break;
		case 64: kernelLinkedListsRegisters<64><<<grid,block>>>(headPtrs, nextPtrs, dataPtr, framebuffer, stride, pixels); break;
		case 128: kernelLinkedListsRegisters<128><<<grid,block>>>(headPtrs, nextPtrs, dataPtr, framebuffer, stride, pixels); break;
		case 256: kernelLinkedListsRegisters<256><<<grid,block>>>(headPtrs, nextPtrs, dataPtr, framebuffer, stride, pixels); break;
		case 512: kernelLinkedListsRegisters<512><<<grid,block>>>(headPtrs, nextPtrs, dataPtr, framebuffer, stride, pixels); break;
		default: success = false; break;
		}
	}
	else if (mergeSort)
	{
		SWITCH_FRAGS(maxFrags, kernelLinkedListsMerge, headPtrs, nextPtrs, dataPtr, framebuffer, stride, pixels);
	}
	else
	{
		switch (maxFrags)
		{
		case 8: kernelLinkedLists<8><<<grid,block>>>(headPtrs, nextPtrs, dataPtr, framebuffer, stride, pixels); break;
		case 16: kernelLinkedLists<16><<<grid,block>>>(headPtrs, nextPtrs, dataPtr, framebuffer, stride, pixels); break;
		case 32: kernelLinkedLists<32><<<grid,block>>>(headPtrs, nextPtrs, dataPtr, framebuffer, stride, pixels); break;
		case 64: kernelLinkedLists<64><<<grid,block>>>(headPtrs, nextPtrs, dataPtr, framebuffer, stride, pixels); break;
		case 128: kernelLinkedLists<128><<<grid,block>>>(headPtrs, nextPtrs, dataPtr, framebuffer, stride, pixels); break;
		case 256: kernelLinkedLists<256><<<grid,block>>>(headPtrs, nextPtrs, dataPtr, framebuffer, stride, pixels); break;
		case 512: kernelLinkedLists<512><<<grid,block>>>(headPtrs, nextPtrs, dataPtr, framebuffer, stride, pixels); break;
		default: success = false; break;
		}
	}
	
	if (CHECK_CUDA_ERROR)
		success = false;
	
	CUDAGLBuffer::unmapAll();
	return success;
}

bool compositeLinearizedShared(int offsets, int data, int outBufferTexture, int pixels, int maxFrags)
{
	int stride = 2;
	unsigned int* offsetsPtr = (unsigned int*)CUDAGLBuffer::get[offsets];
	float* dataPtr = (float*)CUDAGLBuffer::get[data];
	uchar4* framebuffer = (uchar4*)CUDAGLBuffer::get[outBufferTexture];
		
	const int tpb = 32;
	dim3 grid(ceil(pixels, tpb), 1, 1);
	dim3 block(tpb, 1, 1);
	
	int shared = sizeof(float2) * tpb;
	
	bool success = true;
	
	/*
	switch (maxFrags)
	{
	case 8: kernelLinearizedParallelSpawner<8, tpb><<<grid,block,shared>>>(offsetsPtr, dataPtr, framebuffer, stride, pixels); break;
	case 16: kernelLinearizedParallelSpawner<16, tpb><<<grid,block,shared>>>(offsetsPtr, dataPtr, framebuffer, stride, pixels); break;
	case 32: kernelLinearizedParallelSpawner<32, tpb><<<grid,block,shared>>>(offsetsPtr, dataPtr, framebuffer, stride, pixels); break;
	case 64: kernelLinearizedParallelSpawner<64, tpb><<<grid,block,shared>>>(offsetsPtr, dataPtr, framebuffer, stride, pixels); break;
	case 128: kernelLinearizedParallelSpawner<128, tpb><<<grid,block,shared>>>(offsetsPtr, dataPtr, framebuffer, stride, pixels); break;
	case 256: kernelLinearizedParallelSpawner<256, tpb><<<grid,block,shared>>>(offsetsPtr, dataPtr, framebuffer, stride, pixels); break;
	case 512: kernelLinearizedParallelSpawner<512, tpb><<<grid,block,shared>>>(offsetsPtr, dataPtr, framebuffer, stride, pixels); break;
	default: success = false; break;
	}
	*/
	
	if (CHECK_CUDA_ERROR)
		success = false;
	
	CUDAGLBuffer::unmapAll();
	return success;
}

bool compositeLinearizedGlobal(int offsets, int data, int ids, int outBufferTexture, int pixels, int maxFrags)
{
	return false;
}


